#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "common.h"


constexpr int N = 1<<20;
constexpr int Np = 64;
constexpr int FFT1_SIZE = Np;

__device__ __host__ inline 
hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}


__device__ __host__ inline 
hipfftComplex ComplexCMul(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;
    c.x = a.x * b.x + a.y * b.y;
    c.y = - a.x * b.y + a.y * b.x;
    return c;
}

__device__ hipfftComplex window_callback(void *dataIn, size_t offset, void *callerInfo, void *sharedPtr) {
    static __constant__ float chebwin_128[64] = {
    0.000380749937797,0.000878369075872,0.001843297143275,0.003447797656931,0.005956253668122,0.009691337887956,0.015033089879754,0.022413859893738,
    0.032308562430926,0.045219940772657,0.061658868688135,0.082120089136043,0.107054187233385,0.136836986271137,0.171737908447306,0.211889123375446,
    0.257257487088739,0.307621327529376,0.362554042892542,0.421416240431513,0.483357760728313,0.547330423224679,0.612111721499399,0.676339029480229,
    0.738553197603042,0.797249769654733,0.850935485298114,0.898187294620820,0.937710836166540,0.968395244405256,0.989361268614028,1.000000000000000,
    1.000000000000000,0.989361268614028,0.968395244405256,0.937710836166540,0.898187294620820,0.850935485298114,0.797249769654733,0.738553197603042,
    0.676339029480229,0.612111721499399,0.547330423224679,0.483357760728313,0.421416240431513,0.362554042892542,0.307621327529376,0.257257487088739,
    0.211889123375446,0.171737908447306,0.136836986271137,0.107054187233385,0.082120089136043,0.061658868688135,0.045219940772657,0.032308562430926,
    0.022413859893738,0.015033089879754,0.009691337887956,0.005956253668122,0.003447797656931,0.001843297143275,0.000878369075872,0.000380749937797,
    };
    hipfftComplex input = static_cast<hipfftComplex*>(dataIn)[offset];
    // float *filter = static_cast<float*>(callerInfo);  // The scale factor is passed via callerInfo
    hipfftComplex output;
    output.x = input.x * chebwin_128[offset%Np];
    output.y = input.y * chebwin_128[offset%Np];
    return output;
}

__device__ void transpose_callback(void *dataOut, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr) {
    // Down conversion first then conjugate multiplication and transpose
    hipfftComplex *x = static_cast<hipfftComplex*>(callerInfo);
    size_t row = offset / FFT1_SIZE;
    size_t col = (offset + FFT1_SIZE/2) % FFT1_SIZE;// include FFTshift
    float theta = -2.0*M_PI* (int(col) - Np/2) *row/Np;
    hipfftComplex texp = make_hipFloatComplex(cosf(theta), sinf(theta));
    hipfftComplex tempx = x[row + FFT1_SIZE/2];
    hipfftComplex temp = ComplexMul(element, texp);
    temp = ComplexCMul(temp, tempx);
    // transpose
    size_t idx = col * N + row;
    static_cast<hipfftComplex*>(dataOut)[idx] = temp;
}


__device__ 
hipfftCallbackLoadC d_loadCallbackPtr = window_callback; 
__device__ 
hipfftCallbackStoreC d_storeCallbackPtr = transpose_callback;




int main() {

    hipfftHandle plan, plan2;
    size_t work_size, work_size2;
    float milliseconds2;
    hipfftComplex *d_fft1;
    hipfftComplex *d_fft2;
    hipfftComplex *d_data;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int TSIZE  = N * Np;
    // Allocate device memory
    CHECK_ERROR(hipMallocManaged(&d_fft1, sizeof(hipfftComplex) * TSIZE));
    CHECK_ERROR(hipMallocManaged(&d_fft2, sizeof(hipfftComplex) * TSIZE));
    CHECK_ERROR(hipMallocManaged(&d_data, sizeof(hipfftComplex) * (N + Np -1)));
    // Set initial value
    
    for (int i = 0; i < N + Np-1; i++){
        if(i<N){
            d_data[i].x = float(i)/1024;
            d_data[i].y = float(i)/1024;
        }else{
            d_data[i].x = float(0);
            d_data[i].y = float(0);
        }
        // std::cout << "h_data[" << i << "] = " << h_data[i].x << ", " << h_data[i].y<<"j" << std::endl;
    }
    for (int i = 0; i< N ; i++){
        for (int j = 0; j< Np; j++){
            d_fft1[i*Np +j] = d_data[i+j];
            // std::cout << "input[" << i*Np +j << "] = " << h_input[i*Np +j].x << ", " << h_input[i*Np +j].y << "j" <<std::endl;
        }
    }


    // Copy data from host to device
    hipEventRecord(start, 0);

    for (int itr = 0; itr<1; itr++){

    // Create cuFFT plan
    hipfftCreate(&plan);
    hipfftMakePlan1d(plan, Np, HIPFFT_C2C, N, &work_size);
    hipfftCreate(&plan2);
    hipfftMakePlan1d(plan2, N, HIPFFT_C2C, Np, &work_size2);

    // hipEventRecord(start, 0);
    // Preprocessing callback setup
    hipfftCallbackLoadC h_windowCallback;
    CHECK_ERROR(hipMemcpyFromSymbol(&h_windowCallback, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_windowCallback)));
    CHECK_ERROR(hipfftXtSetCallback(plan, (void **)&h_windowCallback, HIPFFT_CB_LD_COMPLEX, nullptr));

    // Postprocessing callback setup
    hipfftCallbackStoreC h_postprocessingCallback;
    CHECK_ERROR(hipMemcpyFromSymbol(&h_postprocessingCallback, HIP_SYMBOL(d_storeCallbackPtr), sizeof(h_postprocessingCallback)));
    CHECK_ERROR(hipfftXtSetCallback(plan, (void **)&h_postprocessingCallback, HIPFFT_CB_ST_COMPLEX, (void **)&d_data));

    // Execute the FFT
    CHECK_ERROR(hipfftExecC2C(plan, d_fft1, d_fft2, HIPFFT_FORWARD));
    CHECK_ERROR(hipfftExecC2C(plan2, d_fft2, d_fft2, HIPFFT_FORWARD));
    }
    // Copy data from device to host
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start, stop);
    std::cout << "Elapsed time: " << milliseconds2 << " ms" << std::endl;

    // Clean up
    hipfftDestroy(plan);
    hipfftDestroy(plan2);
    hipFree(d_fft1);
    hipFree(d_fft2);
    hipFree(d_data);

    std::cout << "----------------FFT executed with callbacks.-------------" << std::endl;
    return 0;
}

// nvcc -ccbin g++ -std=c++17 -arch sm_86 -O3 -dc -m64 -o callback_example.o -c callback_example.cu
// nvcc -ccbin g++ -arch sm_86 -o callback_example callback_example.o -lcufft_static -lculibos


