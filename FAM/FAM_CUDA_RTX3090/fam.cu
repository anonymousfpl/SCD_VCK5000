/******************************************************************************
 * 全 GPU 版示例：  
 *   1) GPU 端生成 1024 个随机复数并归一化  
 *   2) 复制到 2048 点，再零填到 2240 点  
 *   3) 切成 256×32 矩阵 (步长 L=64, 列优先)  
 *   4) 窗口 → FFT+fftshift → DC 系数 → 转置 → CM+32点 FFT
 *
 * 编译示例:
 *   nvcc -ccbin g++ -std=c++17 -arch=sm_70 -O3 \
 *        -o fam fam.cu -lcufft -lcudart -lcurand
 ******************************************************************************/
//nvcc -ccbin g++ -std=c++17 -arch=sm_70 -O3 -o bin/fam fam.cu -lcufft -lcudart -lcurand
//./bin/fam
//nvprof bin/fam
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand/hiprand.h>
#include <math.h>

// 错误检查宏
#define CHECK_CUDA_ERROR(call)                                \
    do {                                                      \
        hipError_t err = call;                              \
        if (err != hipSuccess) {                            \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) \
                      << " at " << __FILE__ << ":" << __LINE__ \
                      << std::endl;                          \
            exit(EXIT_FAILURE);                              \
        }                                                     \
    } while(0)

#define CHECK_CUFFT_ERROR(call)                               \
    do {                                                      \
        hipfftResult err = call;                              \
        if (err != HIPFFT_SUCCESS) {                          \
            std::cerr << "CUFFT Error: " << err              \
                      << " at " << __FILE__ << ":" << __LINE__ \
                      << std::endl;                          \
            exit(EXIT_FAILURE);                              \
        }                                                     \
    } while(0)

#define CHECK_CURAND_ERROR(call)                              \
    do {                                                      \
        hiprandStatus_t err = call;                           \
        if (err != HIPRAND_STATUS_SUCCESS) {                   \
            std::cerr << "cuRAND Error: " << err             \
                      << " at " << __FILE__ << ":" << __LINE__ \
                      << std::endl;                          \
            exit(EXIT_FAILURE);                              \
        }                                                     \
    } while(0)

// --------------------------- 全局常量 --------------------------- //
static const int origLen = 1024;                 // 原始点数
static const int doubleLen = origLen * 2;        // 2048
static const int P = 32;                         // 矩阵列数
static const int L = 64;                         // 步长
static const int Np = 256;                       // 每列长度
static const int NN = (P - 1) * L + Np;           // 2240 = 31*64 + 256

// --------------------------- 复数运算 --------------------------- //
__device__ hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// 在 CM 步骤里可能需要 conj(a)*b，保留一个示例
__device__ __host__ inline
hipfftComplex ComplexCMul(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;
    c.x = a.x * b.x + a.y * b.y;   
    c.y = - a.x * b.y + a.y * b.x;
    return c;
}

// --------------------------- DC 系数，保存在常量内存 --------------------------- //
alignas(32) __constant__ hipfftComplex dc_coef1[4] = {
    {1.0f, 0.0f}, {1.0f, 0.0f}, {1.0f, 0.0f}, {1.0f, 0.0f}
};
alignas(32) __constant__ hipfftComplex dc_coef2[4] = {
    {1.0f, 0.0f}, {0.0f, -1.0f}, {-1.0f, 0.0f}, {0.0f, 1.0f}
};
alignas(32) __constant__ hipfftComplex dc_coef3[4] = {
    {1.0f, 0.0f}, {-1.0f,0.0f}, {1.0f, 0.0f}, {-1.0f,0.0f}
};
alignas(32) __constant__ hipfftComplex dc_coef4[4] = {
    {1.0f, 0.0f}, {0.0f, 1.0f}, {-1.0f,0.0f}, {0.0f, -1.0f}
};

// --------------------------- (A) 生成 1024 随机复数并归一化 --------------------------- //
// 1) 先用 cuRAND 生成 2*origLen 个 [0,1) 浮点，转为 [-1,1] 存到 d_x
__global__ void genComplexInMinusOneToOne(const float* __restrict__ d_rand,
                                          hipfftComplex* __restrict__ d_x,
                                          int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        float r  = 2.f*(d_rand[2*idx]  - 0.5f); // [-1,1]
        float im = 2.f*(d_rand[2*idx+1]- 0.5f);
        d_x[idx] = make_hipFloatComplex(r, im);
    }
}

// 2) 幅度平方
__global__ void computeMagSquared(const hipfftComplex* __restrict__ d_in,
                                  float* __restrict__ d_out, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        hipfftComplex c = d_in[idx];
        d_out[idx] = c.x*c.x + c.y*c.y;
    }
}

// 3) reduceMax 简单实现：块内归约 => 写回 d_data[blockIdx.x]
__global__ void reduceMax(float* d_data, int n)
{
    extern __shared__ float sdata[]; // 动态共享内存
    int tid = threadIdx.x;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < n) {
        sdata[tid] = d_data[idx];
    } else {
        sdata[tid] = 0.0f;
    }
    __syncthreads();

    for(int stride = blockDim.x/2; stride>0; stride >>= 1){
        if(tid < stride){
            if(sdata[tid] < sdata[tid+stride]){
                sdata[tid] = sdata[tid+stride];
            }
        }
        __syncthreads();
    }

    if(tid==0){
        d_data[blockIdx.x] = sdata[0];
    }
}

// 4) 归一化
__global__ void normalizeComplex(hipfftComplex* d_x, float maxAmp, float Amp, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n){
        hipfftComplex c = d_x[idx];
        d_x[idx].x = Amp*(c.x / maxAmp);
        d_x[idx].y = Amp*(c.y / maxAmp);
    }
}

// --------------------------- (B) 复制 => zero-pad => 构建 256×32 矩阵 --------------------------- //
// 1) 复制: d_out[:n]=d_in, d_out[n:2n]=d_in
__global__ void replicateSignal(const hipfftComplex* d_in, hipfftComplex* d_out, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n){
        d_out[idx]   = d_in[idx];
        d_out[idx+n] = d_in[idx];
    }
}

// 2) zero-pad 到 NN=2240
__global__ void zeroPad(const hipfftComplex* d_in, hipfftComplex* d_out,
                        int oldLen, int newLen)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx<newLen){
        if(idx<oldLen)
            d_out[idx]=d_in[idx];
        else
            d_out[idx]=make_hipFloatComplex(0.f,0.f);
    }
}

// 3) 根据步长 L=64, 每列 256 点，构建 256×32
__global__ void buildOverlapMatrix(const hipfftComplex* __restrict__ d_src,
                                   hipfftComplex* __restrict__ d_matrix,
                                   int rows, int L, int P, int totalLen)
{
    // rows=256, P=32
    int col = blockIdx.x;   // [0..31]
    int row = threadIdx.x;  // [0..255]
    if(col<P && row<rows){
        int start = col * L;
        int idxDest = col*rows + row; // 列优先
        if(start+row < totalLen){
            d_matrix[idxDest] = d_src[start+row];
        } else {
            d_matrix[idxDest] = make_hipFloatComplex(0.f,0.f);
        }
    }
}

// --------------------------- (C) Window, FFT+shift, DC, 转置, CM+32点FFT --------------------------- //
// （以下逻辑保留你原来的代码）

// 窗口 kernel
__global__ void windowingKernel(hipfftComplex *d_matrix, const float *d_window, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    if (row < rows && col < cols) {
        int index = col * rows + row;
        float w = d_window[row];
        d_matrix[index].x *= w;
        d_matrix[index].y *= w;
    }
}
void applyWindow(hipfftComplex *d_matrix, const float *d_window, int rows, int cols)
{
    dim3 blockDim(16, 16);
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x,
                 (rows + blockDim.y - 1) / blockDim.y);
    windowingKernel<<<gridDim, blockDim>>>(d_matrix, d_window, rows, cols);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

// FFT + fftshift
__global__ void fftShiftKernelSimple(hipfftComplex* in, hipfftComplex* out, int rows, int cols) {
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int total  = rows * cols;
    if (offset < total) {
        int col = offset / rows;
        int row = offset % rows;
        int newRow = (row + (rows / 2)) % rows;
        int destIdx = col * rows + newRow;
        out[destIdx] = in[offset];
    }
}
void performFFT(hipfftComplex *d_matrix, int rows, int cols)
{
    hipfftHandle plan;
    int n[1] = { rows };
    CHECK_CUFFT_ERROR(hipfftPlanMany(&plan, 1, n,
                      nullptr, 1, rows,
                      nullptr, 1, rows,
                      HIPFFT_C2C, cols));
    CHECK_CUFFT_ERROR(hipfftExecC2C(plan, d_matrix, d_matrix, HIPFFT_FORWARD));
    CHECK_CUFFT_ERROR(hipfftDestroy(plan));

    // fftshift
    size_t bytes = rows * cols * sizeof(hipfftComplex);
    hipfftComplex* d_temp = nullptr;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_temp, bytes));

    int total = rows * cols;
    int blockSize = 256;
    int gridSize = (total + blockSize - 1) / blockSize;
    fftShiftKernelSimple<<<gridSize, blockSize>>>(d_matrix, d_temp, rows, cols);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    CHECK_CUDA_ERROR(hipMemcpy(d_matrix, d_temp, bytes, hipMemcpyDeviceToDevice));
    CHECK_CUDA_ERROR(hipFree(d_temp));
}

// DC 系数乘法
__global__ void applyDCCoefKernel(hipfftComplex *d_matrix, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    if (row < rows && col < cols) {
        int idx = col * rows + row;
        int patternIdx = row % 4;    
        int coefCol    = col % 4;    

        hipfftComplex coef;
        if (coefCol == 0) {
            coef = dc_coef1[patternIdx];
        } else if (coefCol == 1) {
            coef = dc_coef2[patternIdx];
        } else if (coefCol == 2) {
            coef = dc_coef3[patternIdx];
        } else {
            coef = dc_coef4[patternIdx];
        }
        d_matrix[idx] = ComplexMul(d_matrix[idx], coef);
    }
}
void applyDCCoef(hipfftComplex *d_matrix, int rows, int cols)
{
    dim3 blockDim(16, 16);
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x,
                 (rows + blockDim.y - 1) / blockDim.y);
    applyDCCoefKernel<<<gridDim, blockDim>>>(d_matrix, rows, cols);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

// 转置
__global__ void transposeKernel(const hipfftComplex *in, hipfftComplex *out,
                                int oldRows, int oldCols)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y; 
    if (i < oldCols && j < oldRows) {
        int idx_in  = i * oldRows + j; 
        int idx_out = j * oldCols + i; 
        out[idx_out] = in[idx_in];
    }
}
void transposeMatrix(hipfftComplex *d_in, hipfftComplex *d_out, int oldRows, int oldCols)
{
    dim3 blockDim(16, 16);
    dim3 gridDim((oldCols + blockDim.x - 1) / blockDim.x,
                 (oldRows + blockDim.y - 1) / blockDim.y);
    transposeKernel<<<gridDim, blockDim>>>(d_in, d_out, oldRows, oldCols);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

// CM+32点 FFT
__global__ void cmMulKernel(const hipfftComplex* d_transposed, hipfftComplex* d_cmIn)
{
    int pos    = threadIdx.x;
    int target = blockIdx.x;
    int ref    = blockIdx.y;

    int transformId = ref * 256 + target;
    int outIndex    = transformId * 32 + pos;

    int refIndex    = ref    * 32 + pos;
    int targetIndex = target * 32 + pos;

    hipfftComplex a = d_transposed[refIndex];
    hipfftComplex b = d_transposed[targetIndex];
    hipfftComplex a_conj = make_hipFloatComplex(a.x, -a.y);

    d_cmIn[outIndex] = ComplexMul(a_conj, b);
}
void performCM32ptFFT(const hipfftComplex* d_transposed, hipfftComplex** d_cmFFTResult)
{
    int numTransforms = 256 * 256;
    int fftSize       = 32;
    size_t totalBytes = numTransforms * fftSize * sizeof(hipfftComplex);

    CHECK_CUDA_ERROR(hipMalloc((void**)d_cmFFTResult, totalBytes));

    dim3 gridDim(256, 256);
    dim3 blockDim(32);
    cmMulKernel<<<gridDim, blockDim>>>(d_transposed, *d_cmFFTResult);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    hipfftHandle plan;
    int n[1] = { fftSize };
    CHECK_CUFFT_ERROR(hipfftPlanMany(&plan, 1, n,
                      nullptr, 1, fftSize,
                      nullptr, 1, fftSize,
                      HIPFFT_C2C, numTransforms));
    CHECK_CUFFT_ERROR(hipfftExecC2C(plan, *d_cmFFTResult, *d_cmFFTResult, HIPFFT_FORWARD));
    CHECK_CUFFT_ERROR(hipfftDestroy(plan));
}

// ============================================================================
// (D) 对 65536×32 个复数结果做幅度平方 (abs^2)
// ============================================================================
__global__ void computeAbsSquareKernel(const hipfftComplex* d_in, float* d_out, int total)
{
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx<total){
        hipfftComplex c = d_in[idx];
        d_out[idx] = c.x*c.x + c.y*c.y; // abs^2
    }
}

// 对外接口：
//   d_in  : CM+32pt FFT 完成后得到的数组指针
//   d_out : 输出 float 数组指针 (本函数会在 GPU 上自行分配)
//   N     : 数组长度(65536×32)
void computeAbsSquareAll(const hipfftComplex* d_in, float** d_out, int N)
{
    // 1) 在 GPU 上分配存放 abs^2 结果的 float 数组
    CHECK_CUDA_ERROR(hipMalloc((void**)d_out, N*sizeof(float)));

    // 2) 每个线程处理 1 个复数 => 计算 abs^2
    dim3 block(256);
    dim3 grid((N+block.x-1)/block.x);
    computeAbsSquareKernel<<<grid, block>>>(d_in, *d_out, N);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}


// --------------------------- 主函数 --------------------------- //
int main()
{
    // 创建 CUDA event 用于计时
    hipEvent_t startEvent, stopEvent;
    CHECK_CUDA_ERROR(hipEventCreate(&startEvent));
    CHECK_CUDA_ERROR(hipEventCreate(&stopEvent));
    float elapsedTime = 0.f;
    
    // 在进入迭代前记录整个流程的起始时间
    CHECK_CUDA_ERROR(hipEventRecord(startEvent, 0));

    // 循环 10 次，重复整个处理流程
    for (int iter = 0; iter < 1000; iter++) {
        std::cout << "Iteration " << iter << std::endl;
        
        // --------------------------- (A) 生成 1024 随机复数并归一化 ---------------------------
        // 1) 分配 1024 个复数
        hipfftComplex* d_x = nullptr;
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_x, origLen * sizeof(hipfftComplex)));

        // 2) 分配 2*origLen 个浮点数，生成随机数
        float* d_rand = nullptr;
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_rand, 2 * origLen * sizeof(float)));

        hiprandGenerator_t gen;
        CHECK_CURAND_ERROR(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        CHECK_CURAND_ERROR(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
        CHECK_CURAND_ERROR(hiprandGenerateUniform(gen, d_rand, 2 * origLen));

        // 3) 将 [0,1) 转换到 [-1,1]，生成复数存入 d_x
        {
            dim3 block(256);
            dim3 grid((origLen + block.x - 1) / block.x);
            genComplexInMinusOneToOne<<<grid, block>>>(d_rand, d_x, origLen);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
        }
        CHECK_CUDA_ERROR(hipFree(d_rand));
        CHECK_CURAND_ERROR(hiprandDestroyGenerator(gen));

        // 4) 计算幅度平方，归约求最大值并归一化
        float* d_mag2 = nullptr;
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_mag2, origLen * sizeof(float)));
        {
            dim3 block(256);
            dim3 grid((origLen + block.x - 1) / block.x);
            computeMagSquared<<<grid, block>>>(d_x, d_mag2, origLen);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
        }
        int currSize = origLen;
        int blockSize = 256;
        while (currSize > 1) {
            int gridSize = (currSize + blockSize - 1) / blockSize;
            reduceMax<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_mag2, currSize);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
            currSize = gridSize;
        }
        float maxVal2;
        CHECK_CUDA_ERROR(hipMemcpy(&maxVal2, d_mag2, sizeof(float), hipMemcpyDeviceToHost));
        float maxVal = sqrtf(maxVal2);

        {
            float Amp = 1.0f; // 放大因子
            dim3 block(256);
            dim3 grid((origLen + block.x - 1) / block.x);
            normalizeComplex<<<grid, block>>>(d_x, maxVal, Amp, origLen);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
        }
        CHECK_CUDA_ERROR(hipFree(d_mag2));

        // --------------------------- (B) 复制 => zero-pad => 构建 256×32 矩阵 ---------------------------
        // 1) 复制到 2048（即把原始信号复制一遍）
        hipfftComplex* d_x2 = nullptr;
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_x2, doubleLen * sizeof(hipfftComplex)));
        {
            dim3 block(256);
            dim3 grid((origLen + block.x - 1) / block.x);
            replicateSignal<<<grid, block>>>(d_x, d_x2, origLen);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
        }
        CHECK_CUDA_ERROR(hipFree(d_x)); // 已复制完

        // 2) zero-pad 到 2240 点
        hipfftComplex* d_xx = nullptr;
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_xx, NN * sizeof(hipfftComplex)));
        {
            dim3 block(256);
            dim3 grid((NN + block.x - 1) / block.x);
            zeroPad<<<grid, block>>>(d_x2, d_xx, doubleLen, NN);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
        }
        CHECK_CUDA_ERROR(hipFree(d_x2));

        // 3) 构造 256×32 矩阵（列优先，每列256点）
        hipfftComplex* d_matrix = nullptr;
        size_t matrixBytes = Np * P * sizeof(hipfftComplex);
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_matrix, matrixBytes));
        {
            // 每个 block 负责一列，block 内 256 线程对应矩阵行
            dim3 block(Np);
            dim3 grid(P);
            buildOverlapMatrix<<<grid, block>>>(d_xx, d_matrix, Np, L, P, NN);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
        }
        CHECK_CUDA_ERROR(hipFree(d_xx));

        // --------------------------- (C) 后续处理：窗口 → FFT+fftshift → DC → 转置 → CM+32点 FFT → abs^2 ---------------------------
        // (C1) 窗口处理：将主机窗口数据拷贝到设备后调用核函数
        float h_window[Np] = {
            0.079999924,0.08013916,0.080558777,0.081256866,0.082231522,0.083486557,0.085018158,
            0.086826324,0.088907242,0.091264725,0.093893051,0.096792221,0.099962234,0.10339737,
            0.10709953,0.111063,0.11528778,0.11976814,0.124506,0.12949562,0.13473511,0.14021873,
            0.1459465,0.15191269,0.15811539,0.16454887,0.17121124,0.17809677,0.18520355,
            0.19252396,0.20005608,0.20779419,0.21573448,0.22387123,0.23220062,0.24071503,
            0.24941254,0.25828743,0.26733208,0.27654266,0.28591156,0.29543686,0.30510902,
            0.31492615,0.32487679,0.33496094,0.34516716,0.35549355,0.36593056,0.37647438,
            0.38711739,0.39785194,0.40867424,0.41957474,0.43054962,0.44159126,0.45269203,
            0.4638443,0.47504425,0.48628426,0.49755669,0.50885391,0.52017021,0.53149986,
            0.54283333,0.55416489,0.56548882,0.57679749,0.58808327,0.59934044,0.61055946,
            0.62173843,0.63286591,0.64393806,0.65494537,0.66588593,0.67674828,0.6875267,
            0.69821739,0.70881081,0.71930122,0.72968483,0.73995209,0.75009727,0.76011467,
            0.77000046,0.7797451,0.78934479,0.79879189,0.80808449,0.81721115,0.82617188,
            0.83495903,0.84356499,0.85198784,0.86022186,0.86826134,0.87610054,0.88373566,
            0.89116287,0.89837646,0.90537262,0.91214752,0.91869545,0.9250145,0.93109894,
            0.93694687,0.94255447,0.94791603,0.95302963,0.95789337,0.96250343,0.96685791,
            0.97095108,0.97478485,0.9783535,0.98165512,0.98468971,0.98745537,0.98994827,
            0.99216843,0.99411201,0.99578285,0.99717522,0.99829102,0.99912834,0.99968529,
            0.99996567,0.99996567,0.99968529,0.99912834,0.99829102,0.99717522,0.99578285,
            0.99411201,0.99216843,0.98994827,0.98745537,0.98468971,0.98165512,0.9783535,
            0.97478485,0.97095108,0.96685791,0.96250343,0.95789337,0.95302963,0.94791603,
            0.94255447,0.93694687,0.93109894,0.9250145,0.91869545,0.91214752,0.90537262,
            0.89837646,0.89116287,0.88373566,0.87610054,0.86826134,0.86022186,0.85198784,
            0.84356499,0.83495903,0.82617188,0.81721115,0.80808449,0.79879189,0.78934479,
            0.7797451,0.77000046,0.76011467,0.75009727,0.73995209,0.72968483,0.71930122,
            0.70881081,0.69821739,0.6875267,0.67674828,0.66588593,0.65494537,0.64393806,
            0.63286591,0.62173843,0.61055946,0.59934044,0.58808327,0.57679749,0.56548882,
            0.55416489,0.54283333,0.53149986,0.52017021,0.50885391,0.49755669,0.48628426,
            0.47504425,0.4638443,0.45269203,0.44159126,0.43054962,0.41957474,0.40867424,
            0.39785194,0.38711739,0.37647438,0.36593056,0.35549355,0.34516716,0.33496094,
            0.32487679,0.31492615,0.30510902,0.29543686,0.28591156,0.27654266,0.26733208,
            0.25828743,0.24941254,0.24071503,0.23220062,0.22387123,0.21573448,0.20779419,
            0.20005608,0.19252396,0.18520355,0.17809677,0.17121124,0.16454887,0.15811539,
            0.15191269,0.1459465,0.14021873,0.13473511,0.12949562,0.124506,0.11976814,
            0.11528778,0.111063,0.10709953,0.10339737,0.099962234,0.096792221,0.093893051,
            0.091264725,0.088907242,0.086826324,0.085018158,0.083486557,0.082231522,
            0.081256866,0.080558777,0.08013916,0.079999924
        };
        float* d_window = nullptr;
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_window, Np * sizeof(float)));
        CHECK_CUDA_ERROR(hipMemcpy(d_window, h_window, Np * sizeof(float), hipMemcpyHostToDevice));
        applyWindow(d_matrix, d_window, Np, P);
        CHECK_CUDA_ERROR(hipFree(d_window));

        // (C2) FFT + fftshift
        performFFT(d_matrix, Np, P);

        // (C3) DC 系数乘法
        applyDCCoef(d_matrix, Np, P);

        // (C4) 转置 => (32×256)
        hipfftComplex* d_transposed = nullptr;
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_transposed, matrixBytes));
        transposeMatrix(d_matrix, d_transposed, Np, P);

        // (C5) CM+32点 FFT
        hipfftComplex* d_cmFFTResult = nullptr;
        performCM32ptFFT(d_transposed, &d_cmFFTResult);
        CHECK_CUDA_ERROR(hipFree(d_transposed));

        // (D) 对 d_cmFFTResult 进行 abs^2 计算，得到 65536×32 个结果
        int totalCM = 256 * 256 * 32;
        float* d_absSquare = nullptr;
        computeAbsSquareAll(d_cmFFTResult, &d_absSquare, totalCM);

        // ============ 拷回部分结果进行检查 ============
        const int checkTransforms = 4;
        const int checkCount = checkTransforms * 32;
        hipfftComplex* h_cmFFTResult = (hipfftComplex*)malloc(checkCount * sizeof(hipfftComplex));
        CHECK_CUDA_ERROR(hipMemcpy(h_cmFFTResult, d_cmFFTResult,
                                    checkCount * sizeof(hipfftComplex),
                                    hipMemcpyDeviceToHost));

        float* h_absSquare = (float*)malloc(checkCount * sizeof(float));
        CHECK_CUDA_ERROR(hipMemcpy(h_absSquare, d_absSquare,
                                    checkCount * sizeof(float),
                                    hipMemcpyDeviceToHost));
        std::cout << "\n前 " << checkTransforms << " 路(共 " << checkCount << " 点) 的 abs^2 数据(前 8 个):\n";
        for (int i = 0; i < 8; i++) {
            std::cout << " " << h_absSquare[i];
        }
        std::cout << "\n";

        // 释放本次迭代分配的所有资源
        free(h_cmFFTResult);
        free(h_absSquare);
        CHECK_CUDA_ERROR(hipFree(d_absSquare));
        CHECK_CUDA_ERROR(hipFree(d_matrix));
        CHECK_CUDA_ERROR(hipFree(d_cmFFTResult));
    }
    
    // 在所有迭代结束后记录结束时间
    CHECK_CUDA_ERROR(hipEventRecord(stopEvent, 0));
    CHECK_CUDA_ERROR(hipEventSynchronize(stopEvent));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));
    std::cout << "[GPU] Total processing time: " << elapsedTime << " ms\n";
    
    // 清理计时用的 CUDA event
    CHECK_CUDA_ERROR(hipEventDestroy(startEvent));
    CHECK_CUDA_ERROR(hipEventDestroy(stopEvent));
    
    return 0;
}
